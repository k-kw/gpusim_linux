#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

#include "my_all.h"
#include "complex_array_class.h"
#include "dvcfnc.cuh"





//乱数ライブラリインクルード
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



using namespace std;

//CUDA
#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))


//関数群

////template of under function
//template <class Type>
//__global__ void cusetcucomplex(hipComplex* com, Type* Re, Type* Im, int size)
//{
//
//    int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//    if (idx < size) {
//        com[idx] = make_hipComplex((float)Re[idx], (float)Im[idx]);
//    }
//}


//double to hipComplex
__global__ void cusetcucomplex(hipComplex* com, double* Re, double* Im, int size)
{

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < size) {
        com[idx] = make_hipComplex((float)Re[idx], (float)Im[idx]);
    }
}

// unsigned char to hipComplex
__global__ void uc2cucomplex(hipComplex* com, unsigned char* Re, int size)
{

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < size) {
        com[idx] = make_hipComplex((float)Re[idx], 0.0f);
    }
}



//normalization after fft
__global__ void normfft(hipfftComplex* dev, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < x * y) {
        dev[idx] = make_hipComplex(hipCrealf(dev[idx]) / (x * y), hipCimagf(dev[idx]) / (x * y));
    }
}


//2D fft
void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;

    //hipfftPlan2d 第2引数 : 最も遅く変化する次元のサイズ
    //hipfftPlan2d 第3引数 : 最も速く変化する次元のサイズ
    hipfftPlan2d(&plan, y, x, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}

//2d inverse fft
void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;

    //hipfftPlan2d 第2引数 : 最も遅く変化する次元のサイズ
    //hipfftPlan2d 第3引数 : 最も速く変化する次元のサイズ
    hipfftPlan2d(&plan, y, x, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
    hipfftDestroy(plan);
}

//cufftcomplex to My_ComArray
void cufftcom2mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
    for (int i = 0; i < s; i++) {
        out->Re[i] = (double)hipCrealf(in[i]);
        out->Im[i] = (double)hipCimagf(in[i]);

    }
}


//make angular spectrum method's H 
__global__ void Hcudaf(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {
        Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
        Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
    }
}

//make angular spectrum method's H (hipComplex)
__global__ void HcudacuCom(hipComplex* H, int x, int y, float z, float d, float lam)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    float u = 1 / (x * d), v = 1 / (y * d);


    if (idy < y && idx < x) {
        H[idy * x + idx] = make_hipComplex(cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2)))),
            sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2)))));
    }
}

__global__ void  shiftf(float* ore, float* oim, float* re, float* im, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {

        if (idx < x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
            ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
            oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
        }
        else if (idx >= x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
            ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
            oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
        }
    }
}

//use
__global__ void shiftCom(hipComplex* out, hipComplex* in, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {

        if (idx < x / 2 && idy < y / 2) {
            out[idy * x + idx] = in[(idy + y / 2) * x + (idx + x / 2)];
            out[(idy + y / 2) * x + (idx + x / 2)] = in[idy * x + idx];

        }
        else if (idx >= x / 2 && idy < y / 2) {
            out[idy * x + idx] = in[(idy + y / 2) * x + (idx - x / 2)];
            out[(idy + y / 2) * x + (idx - x / 2)] = in[idy * x + idx];

        }
    }
}


//floatXcufftCom
__global__ void mulcomcufftcom(hipfftComplex* out, float* re, float* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex(re[idx] * hipCrealf(in[idx]) - im[idx] * hipCimagf(in[idx]),
            re[idx] * hipCimagf(in[idx]) + im[idx] * hipCrealf(in[idx]));

    }
}


//doubleXcufftCom
__global__ void muldoublecomcufftcom(hipfftComplex* out, double* re, double* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex((float)re[idx] * hipCrealf(in[idx]) - (float)im[idx] * hipCimagf(in[idx]),
            (float)re[idx] * hipCimagf(in[idx]) + (float)im[idx] * hipCrealf(in[idx]));

    }
}


//use
__global__ void Cmulfft(hipComplex* out, hipComplex* fin, hipComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //レジスタver
    //hipComplex tmp1, tmp2;
    

    if (idx < s) {
        /*tmp1 = make_hipComplex(hipCrealf(fin[idx]), hipCimagf(fin[idx]));
        tmp2 = make_hipComplex(hipCrealf(in[idx]), hipCimagf(in[idx]));*/
        //out[idx] = hipCmulf(tmp1, tmp2);

        out[idx] = hipCmulf(fin[idx], in[idx]);

    }

}

//use
__global__ void pad_cufftcom2cufftcom(hipfftComplex* out, int lx, int ly, hipfftComplex* in, int sx, int sy)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[(idy + ly / 4) * lx + (idx + lx / 4)] = in[idy * sx + idx];
    }

}

__global__ void elimpad(hipfftComplex* out, int sx, int sy, hipfftComplex* in, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[idy * sx + idx] = in[(idy + ly / 4) * lx + (idx + lx / 4)];
    }
}

__global__ void elimpad2Cmulfft(hipComplex* outmlt, hipComplex* opponent, 
    int sx, int sy, hipComplex* in, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    hipComplex tmp1, tmp2;

    if (idx < sx && idy < sy) {
        //真ん中を取り出す
        tmp1 = make_hipComplex(hipCrealf(in[(idy + ly / 4) * lx + (idx + lx / 4)]), 
            hipCimagf(in[(idy + ly / 4) * lx + (idx + lx / 4)]));

        //レンズ配列等
        tmp2 = make_hipComplex(hipCrealf(opponent[idy * sx + idx]), hipCimagf(opponent[idy * sx + idx]));

        outmlt[idy * sx + idx] = hipCmulf(tmp1, tmp2);
    }

}


void Hcudaf_shiftf(float* devReH, float* devImH, int x, int y, float d, float z, float lamda, dim3 grid, dim3 block) {
    float* ReH, * ImH;
    hipMalloc((void**)&ReH, sizeof(float) * x * y);
    hipMalloc((void**)&ImH, sizeof(float) * x * y);

    float u = 1 / (x * d), v = 1 / (y * d);

    Hcudaf << <grid, block >> > (ReH, ImH, x, y, u, v, z, lamda);
    shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);

    hipFree(ReH);
    hipFree(ImH);
}

//use
void Hcudashiftcom(hipComplex* dev, int x, int y, float z, float d, float lamda, dim3 grid, dim3 block) {
    hipComplex* tmp;
    hipMalloc((void**)&tmp, sizeof(hipComplex) * x * y);

    HcudacuCom << <grid, block >> > (tmp, x, y, z, d, lamda);
    shiftCom << <grid, block >> > (dev, tmp, x, y);

    hipFree(tmp);

}


__global__ void cucompower(double* power, hipComplex* dev, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        power[idx] = sqrt((double)sqr(hipCrealf(dev[idx])) + (double)sqr(hipCimagf(dev[idx])));

    }
}

//use
__global__ void elimpadcucompower(double* power ,int sx, int sy, hipComplex* dev, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    //レジスタver
    double tmp1, tmp2;

    if (idx < sx && idy < sy) {
        tmp1 = (double)sqr(hipCrealf(dev[(idy + ly / 4) * lx + (idx + lx / 4)]));
        tmp2 = (double)sqr(hipCimagf(dev[(idy + ly / 4) * lx + (idx + lx / 4)]));
        power[idy * sx + idx] = sqrt( tmp1 + tmp2 );
    }
}



//use
__global__ void cunormaliphase(hipComplex* out, double* normali, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex((float)cos(2 * M_PI * normali[idx]), (float)sin(2 * M_PI * normali[idx]));

    }

}


