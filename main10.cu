#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include "my_all.h"
#include "Bmp_class.h"
#include "complex_array_class.h"
#include "dvcfnc.cuh"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

//copy
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//乱数ライブラリインクルード
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
//copy


using namespace std;
using namespace cv;

//パラメータ
#define BX 28       //bindat横
#define BY 28       //bindatの縦

//SLMに合わせたほうがいい？

//#define SX 4096     //SLMでの横画素数(4で割れる整数に限る)
//#define SY 2400     //SLMでの縦画素数(4で割れる整数に限る)
//float d = 3.74e-06;
//#define short 2400    //短辺

//SLM解像度
#define SLMX 4096     //SLMでの横画素数(4で割れる整数に限る)
#define SLMY 2400     //SLMでの縦画素数(4で割れる整数に限る)
#define short 2400    //短辺
#define SLMSIZE (SLMX*SLMY)

//シミュレーション配列サイズ
#define SX 8192
#define SY 4800
#define SIZE (SX*SY)      //パディング前サイズ

//SX,SYの画素ピッチ
float d = 1.87e-06;

//0埋め後画像サイズ
#define SX2 (2*SX)
#define SY2 (2*SY)
#define PADSIZE (SX2*SY2) //パディング後サイズ

#define N 70000       //画像の枚数
#define CHECK_NUM N  //シミュレーション画像をチェックする番号

//#define lam 532e-09  //波長
//#define d 1.496e-05 //画素ピッチ
//#define a 0.1 //伝搬距離1
//#define b 0.03 //伝搬距離2
//#define f 0.03 //焦点距離

//波長
float lamda = 532e-09;

//レンズ拡散版の寸法とSLMから決める
//#define LENS_SIZE 32 //拡散板レンズのレンズサイズ
//1mm(レンズ角)/d(SLMピッチ)=267より
#define LENS_SIZE 512

//伝搬距離と焦点距離
float a = 0.04;
//float b = 0.03;
float b = 0.04;
//float f = 0.001;
//フライアイレンズのデータシートより
float f = 0.0033;

////NEW
////SLM解像度に対する、カメラの解像度の割合
//#define SC 0.5
////カメラの解像度
//#define CAMX (int)(SX*SC)
//#define CAMY (int)(SY*SC)
////NEW

#define resolution pow(2, 8) //解像度
#define approx false    //レンズの式の近似
#define sqr(x) ((x)*(x))

//copy
//CUDA
#ifndef __HIPCC__
#define __HIPCC__
#endif 
//copy


//1次元のグリッドとブロック
//総スレッド数
// PADSIZEに同じ
//#define Nthread SX2*SY2
// 
//ブロック内のスレッド数1=<BS=<1024
#define BS 1024



//二次元のグリッドとブロック
//ブロック当たりのスレッド数は合計1024までなので、block(32,32)より増やせない
//gridは上限ない？
//SX,SY=512,512
//dim3 grid(32, 32), block(32, 32), grid2(16, 16);


//今のところ2の階乗の正方形でしかできない,デバッグの必要あり？
//SX,SY=4096,4096
//dim3 grid(256, 256), block(32, 32), grid2(128, 128);

#define blockx 32
#define blocky 32

dim3 grid((SX2 + blockx - 1) / blockx, (SY2 + blocky - 1) / blocky), block(blockx, blocky), grid2((SX + blockx - 1) / blockx, (SY + blocky - 1) / blocky);

//shared memoryは1ブロックに16KB, floatなら4096個, doubleならその半分


//テンプレート関数だけ別にするとうまくいかない
//テンプレート関数の型まで明示したインスタンスを定義のファイルにおいておけばOK？

//use
template <class Type>
__global__ void cunormali(Type* devin, Type* devout, Type max, Type min, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        devout[idx] = (devin[idx] - min) / (max - min);

    }
}

bool samevalue_sclup(My_ComArray_2D *out, My_ComArray_2D *in) {
    int xml, yml, inx, iny, outx, outy;
    inx = in->x;
    iny = in->y;
    outx = out->x;
    outy = out->y;

    if ((outx <= inx) || (outy <= iny)) {
        cout << "出力配列の幅と高さはいずれも入力より大きくしてください" << endl;
        return false;
    }

    
    xml = (outx + inx - 1) / inx;
    yml = (outy + iny - 1) / iny;
    

    //cout << xml << yml << endl;

    for (int i = 0; i < outy; i++) {
        for (int j = 0; j < outx; j++) {
            out->Re[i * outx + j] = in->Re[(int)(i / yml) * inx + (int)(j / xml)];
            out->Im[i * outx + j] = in->Im[(int)(i / yml) * inx + (int)(j / xml)];

        }
    }
    return true;
}

//
//__global__ void samevl_sclup_cuda(double* out, int outx, int outy, double* in, int inx, int iny) {
//
//    int idx = blockDim.x * blockIdx.x + threadIdx.x;
//    int idy = blockDim.y * blockIdx.y + threadIdx.y;
//
//    int xml, yml, tmpy, tmpx;
//    
//    xml = (outx + inx - 1) / inx;
//    yml = (outy + iny - 1) / iny;
//    tmpy = (int)idy / yml;
//    tmpx = (int)idx / xml;
//
//    if (idx < outx && idy < outy) {
//        out[idy * outx + idx] = in[tmpy * inx + tmpx];
//
//    }
//}

//テンプレート
template <class Type>
__global__ void samevl_sclup_cuda_anytype2double(double* out, int outx, int outy, Type* in, int inx, int iny) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    int xml, yml, tmpy, tmpx;

    xml = (outx + inx - 1) / inx;
    yml = (outy + iny - 1) / iny;
    tmpy = (int)idy / yml;
    tmpx = (int)idx / xml;

    if (idx < outx && idy < outy) {
        out[idy * outx + idx] = (double)in[tmpy * inx + tmpx];

    }
}

//
//__global__ void samevl_sclup_cuda_uc2double(double* out, int outx, int outy, unsigned char* in, int inx, int iny) {
//
//    int idx = blockDim.x * blockIdx.x + threadIdx.x;
//    int idy = blockDim.y * blockIdx.y + threadIdx.y;
//
//    int xml, yml, tmpy, tmpx;
//
//    xml = (outx + inx - 1) / inx;
//    yml = (outy + iny - 1) / iny;
//    tmpy = (int)idy / yml;
//    tmpx = (int)idx / xml;
//
//    if (idx < outx && idy < outy) {
//        out[idy * outx + idx] = (double)in[tmpy * inx + tmpx];
//
//    }
//}

//sx:lxとsy:lyが同じ比率に限る
void sum_scldown(double* out, int sx, int sy, double* in, int lx, int ly) {
    int mul;
    mul = (lx + sx - 1) / sx;

    //初期化
    for (int i = 0; i < sy; i++) {
        for (int j = 0; j < sx; j++) {
            out[i * sx + j] = 0;
        }
    }
    
    for (int i = 0; i < ly; i++) {
        for (int j = 0; j < lx; j++) {
            out[(i / mul) * sx + (j / mul)] += in[i * lx + j];
        }
    }
}

//CUDAでやると出力がおかしい
//出力メモリはhipMemsetで０にしておくべき
__global__ void sum_scldwn_cuda(double* out, int sx, int sy, double* in, int lx, int ly) {
    int mul;
    mul = (lx + sx - 1) / sx;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    //int tmpx, tmpy;

    //__shared__ double tmpsum;

    if (idx < lx && idy < ly) {
        /*tmpx = idx / mul;
        tmpy = idy / mul;*/
        out[(idy / mul) * sx + (idx / mul)] += in[idy * lx + idx];
    }

}


//ファイルパス
string binpath = "../../dat/bindat/1byte/fm_28_1.dat";
string simpath = "../../dat/simdat/SLM_phase/1byte/lsd/fm/test_sim.dat";
string oriimg = "./test.bmp";
string simimg = "./testsim_last.bmp";
string scaledown = "./scdwn_last.bmp";
string oriimgexp = "exp.bmp";
string debug_bflens = "bfrlens.bmp";


int main() {
    clock_t start, lap;
    start = clock();

    //画像データをそのままリサイズするか、正方形のまま拡大後横にパディングするか
    int cf_pad;
    do {
        cout << "\n正方形画像データをそのまま長方形に拡大する場合：0を入力\n正方形で拡大後パディングして長方形とする場合：1を入力\n";
        cout << " 0 or 1 : "; cin >> cf_pad;

    } while (cf_pad != 0 && cf_pad != 1);



    //画像データを振幅情報(実部)とするか、位相に変換するか確認
    int ampl_or_phase;
    do {
        cout << "\n画像データを振幅情報(実部)とする場合：0を入力\t正規化後、位相情報とする場合：1を入力\n";
        cout << " 0 or 1 : "; cin >> ampl_or_phase;

    } while (ampl_or_phase != 0 && ampl_or_phase != 1);



    //ランダム位相かレンズアレイか確認
    int rand_or_lsd;
    do {
        cout << "\n拡散板をランダム位相とする場合：0を入力\t拡散板を微小レンズアレイとする場合：1を入力\n";
        cout << " 0 or 1 : "; cin >> rand_or_lsd;

    } while (rand_or_lsd != 0 && rand_or_lsd != 1);

    //ファイル入力・バイナリストリームオープン
    ifstream ifs(binpath, ios::binary /*| ios::in*/);
    //ファイル出力・バイナリストリームオープン
    ofstream ofs(simpath, ios::binary/* | ios::out*/);



    //両方オープンできたか確認
    if ((ifs) && (ofs)) {

        My_LensArray* Lens;
        My_ComArray_2D* Lenspad;
        Lens = new My_LensArray(SIZE, SX, SY, approx, (double)f, (double)lamda, (double)d);
        Lenspad = new My_ComArray_2D(PADSIZE, SX2, SY2);

        if (rand_or_lsd == 0) {
            //ランダム拡散板
            Lens->diffuser_Random(0);

        }
        else {
            //レンズアレイ拡散板
            Lens->diffuser_Lensarray(LENS_SIZE);

        }
        Lens->zeropad(Lenspad);
        delete Lens;

        //LENS
        double* dvbfdpd, * dvbfdpd2;
        hipMalloc((void**)&dvbfdpd, sizeof(double) * PADSIZE);
        hipMalloc((void**)&dvbfdpd2, sizeof(double) * PADSIZE);
        hipMemcpy(dvbfdpd, Lenspad->Re, sizeof(double) * PADSIZE, hipMemcpyHostToDevice);
        hipMemcpy(dvbfdpd2, Lenspad->Im, sizeof(double) * PADSIZE, hipMemcpyHostToDevice);
        delete Lenspad;

        hipComplex* Ldev;
        hipMalloc((void**)&Ldev, sizeof(hipComplex) * PADSIZE);
        cusetcucomplex<<<(PADSIZE + BS - 1) / BS, BS >>>(Ldev, dvbfdpd, dvbfdpd2, PADSIZE);

        hipFree(dvbfdpd);hipFree(dvbfdpd2);
        //LENS


        unsigned char* dvbfucq;
        hipMalloc((void**)&dvbfucq, sizeof(unsigned char) * SLMSIZE);

        //デバイス、double メモリ
        double* dvbfd, * dvbfd2;
        hipMalloc((void**)&dvbfd, sizeof(double) * SIZE);
        hipMalloc((void**)&dvbfd2, sizeof(double) * SIZE);

        //デバイス,hipfftComplexメモリ
        hipfftComplex* dvbffc;
        hipMalloc((void**)&dvbffc, sizeof(hipfftComplex) * SIZE);


        //デバイス,hipfftComplex,PADSIZEメモリ
        hipfftComplex* dvbffcpd;
        hipMalloc((void**)&dvbffcpd, sizeof(hipfftComplex)* PADSIZE);

        //Hメモリ
        hipComplex* Ha;
        hipMalloc((void**)&Ha, sizeof(hipComplex) * PADSIZE);
        Hcudashiftcom(Ha, SX2, SY2, a, d, lamda, grid, block);
        hipComplex* Hb;
        hipMalloc((void**)&Hb, sizeof(hipComplex) * PADSIZE);
        Hcudashiftcom(Hb, SX2, SY2, b, d, lamda, grid, block);


        //ホスト側ページ固定メモリ
        double* hostbfd;
        hipHostMalloc((void**)&hostbfd, sizeof(double) * SIZE);

        unsigned char* hostbfuc;
        hipHostMalloc((void**)&hostbfuc, sizeof(unsigned char) * SLMSIZE);

        
        //ホスト側通常メモリ
        unsigned char* chRe;
        chRe = new unsigned char[BX * BY];

        double* scldwn, * Pline;
        scldwn = new double[SLMSIZE];
        Pline = new double[SLMX];

        int* intw;
        unsigned char* chw;
        intw = new int[SLMX];
        chw = new unsigned char[SLMX];

        for (int k = 0; k < N; k++) {
            //進捗状況表示
            if (k == 0) {
                cout << "\n\n\n-------------------------------出力ファイル作成中---------------------------------\n\n\n";
            }

            //バイナリ読み込み配列ポインタ
            


            //data読み取り
            //1byteで一枚分読み込み
            ifs.read((char*)chRe, sizeof(unsigned char) * BX * BY);
            //上下反転
            invert_img<unsigned char>(chRe, chRe, BX, BY);

            //画像データ確認
            if (k == N - 1) {

                My_Bmp* check;
                check = new My_Bmp(BX, BY);
                check->uc_to_img(chRe);
                check->img_write(oriimg);
                delete check;
            }

            //画像データを拡大するときCV_8Uでやる
            //画像データをcv::Matにコピー
            Mat bin_mat(BY, BX, CV_8U);
            memcpy(bin_mat.data, chRe, BX * BY * sizeof(unsigned char));
            /*imshow("View", bin_mat);
            waitKey(0);*/

            if (cf_pad == 0) {
                //正方形から長方形に直接リサイズ
                Mat bin_mat_res(SLMX, SLMY, CV_8U);
                resize(bin_mat, bin_mat_res, Size(SLMX, SLMY));
                bin_mat.release();

                //拡大したcv::MatをpadReにコピー
                memcpy(hostbfuc, bin_mat_res.data, SLMSIZE * sizeof(unsigned char));
                bin_mat_res.release();
            }
            else {
                //正方形から正方形へ拡大後、横をパディングして長方形に
                //拡大
                Mat bin_mat_res(short, short, CV_8U);
                resize(bin_mat, bin_mat_res, Size(short, short));
                bin_mat.release();
                /*string resizeimg = "resize.bmp";
                imwrite(resizeimg, bin_mat_res);
                imshow("View", bin_mat_res);
                waitKey(0);*/

                //ゼロ埋めして合わせる
                Mat bin_mat_pjr(SLMY, SLMX, CV_8U);
                copyMakeBorder(bin_mat_res, bin_mat_pjr, (int)(SLMY - short) / 2, (int)(SLMY - short) / 2, (int)(SLMX - short) / 2, (int)(SLMX - short) / 2, BORDER_CONSTANT, 0);
                bin_mat_res.release();
                /*string padimg = "pad.bmp";
                imwrite(padimg, bin_mat_pjr);
                imshow("View", bin_mat_pjr);
                waitKey(0);*/

                //拡大したcv::MatをpadReにコピー
                memcpy(hostbfuc, bin_mat_pjr.data, SLMSIZE * sizeof(unsigned char));
                bin_mat_pjr.release();

            }

            //画像データ確認
            if (k == N - 1) {

                My_Bmp* check;
                check = new My_Bmp(SLMX, SLMY);

                check->uc_to_img(hostbfuc);
                check->img_write(oriimgexp);

                delete check;

            }

            hipMemcpy(dvbfucq, hostbfuc, sizeof(unsigned char) * SLMSIZE, hipMemcpyHostToDevice);
            
            samevl_sclup_cuda_anytype2double<unsigned char><<<grid2, block >>>(dvbfd, SX, SY, dvbfucq, SLMX, SLMY);
            hipMemset(dvbfd2, 0, sizeof(double)* SIZE);
            
            if (ampl_or_phase == 0) {
                //振幅変調
                cusetcucomplex<<<(SIZE + BS - 1) / BS, BS >>>(dvbffc, dvbfd, dvbfd2, SIZE);

            }
            else {
                //位相変調
                double* Remax, * Remin;
                Remax = new double;
                Remin = new double;
                
                *Remax = (double)get_max<unsigned char>(hostbfuc, SLMSIZE);
                *Remin = (double)get_min<unsigned char>(hostbfuc, SLMSIZE);

                
                cunormali<double><<<(SIZE + BS - 1) / BS, BS >>>(dvbfd, dvbfd2, *Remax, *Remin, SIZE);
                cunormaliphase<<<(SIZE + BS - 1) / BS, BS >>>(dvbffc, dvbfd2, SIZE);
                delete Remax; delete Remin;
            }


            //パディング
            hipMemset(dvbffcpd, 0, sizeof(hipfftComplex) * PADSIZE);
            pad_cufftcom2cufftcom<<<grid2, block >>>(dvbffcpd, SX2, SY2, dvbffc, SX, SY);
            //角スペクトル
            fft_2D_cuda_dev(SX2, SY2, dvbffcpd);
            Cmulfft<<<(PADSIZE + BS - 1) / BS, BS >>>(dvbffcpd, dvbffcpd, Ha, PADSIZE);
            ifft_2D_cuda_dev(SX2, SY2, dvbffcpd);
            normfft<<<(PADSIZE + BS - 1) / BS, BS >>>(dvbffcpd, SX2, SY2);

            //デバッグ
            if (k == CHECK_NUM - 1) {
                elimpadcucompower<<<grid2, block >>>(dvbfd, SX, SY, dvbffcpd, SX2, SY2);
                hipMemcpy(hostbfd, dvbfd, sizeof(double) * SIZE, hipMemcpyDeviceToHost);
                My_Bmp* check;
                check = new My_Bmp(SX, SY);

                check->data_to_ucimg(hostbfd);
                check->img_write(debug_bflens);
                delete check;

            }
            //レンズをかける
            Cmulfft<<<(PADSIZE + BS - 1) / BS, BS >>>(dvbffcpd, dvbffcpd, Ldev, PADSIZE);
            //角スペクトル
            fft_2D_cuda_dev(SX2, SY2, dvbffcpd);
            Cmulfft<<<(PADSIZE + BS - 1) / BS, BS >>>(dvbffcpd, dvbffcpd, Hb, PADSIZE);
            ifft_2D_cuda_dev(SX2, SY2, dvbffcpd);
            normfft << <(PADSIZE + BS - 1) / BS, BS >> > (dvbffcpd, SX2, SY2);
            //複素振幅出力
            elimpadcucompower<<<grid2, block >>>(dvbfd, SX, SY, dvbffcpd, SX2, SY2);
            //ホストへ
            hipMemcpy(hostbfd, dvbfd, sizeof(double) * SIZE, hipMemcpyDeviceToHost);

            if (k == CHECK_NUM - 1) {

                My_Bmp* check;
                check = new My_Bmp(SX, SY);
                check->data_to_ucimg(hostbfd);
                check->img_write(simimg);
                delete check;

            }

            //CPUで出力振幅をカメラの解像度くらいまで落とす
            if ((int)(SX / SLMX) != (int)(SY / SLMY)) {
                //同じ比率でないなら終了
                cout << "SLM解像度とシミュレーション配列は縦横同じ比率にしてください。\n";
                return 0;
            }
            //memset(scldwn, 0, sizeof(double) * SLMSIZE);
            //近くの値を足しながら縮小、(0で初期化こみ)
            sum_scldown(scldwn, SLMX, SLMY, hostbfd, SX, SY);
            //デバッグ
            if (k == CHECK_NUM - 1) {

                My_Bmp* check;
                check = new My_Bmp(SLMX, SLMY);

                check->data_to_ucimg(scldwn);
                check->img_write(scaledown);
                delete check;

            }           
            mid_line<double>(scldwn, SLMX, SLMY, Pline);

            //書き込み配列
            
            /*norm_reso_n<double>(Pline, intw, (int)(resolution - 1), SX);*/
            norm_reso_n<double>(Pline, intw, (int)(resolution - 1), SLMX);
            
            //to_uch(intw, SX, chw);
            to_uch(intw, SLMX, chw);

            //書き込み
            ofs.write((char*)chw, sizeof(unsigned char)* SLMX);

            if ((k + 1) % 100 == 0) {
                cout << "-----------------------------------" << k + 1 << "--------------------------------------\n";
                lap = clock();
                cout << setprecision(4) << (double)(lap - start) / CLOCKS_PER_SEC / 60 << "分経過\n\n";

            }
        }
        //delete[]intRe;
        delete[]chRe;
        //delete tmp;
        //delete Complex;
        delete[]scldwn;
        delete[]Pline;
        delete[]intw;
        delete[]chw;
        //delete[]padRe;
        hipFree(hostbfd);
        hipFree(hostbfuc);
        //hipFree(dvbfdq);
        hipFree(dvbfucq);
        hipFree(dvbffc);
        hipFree(dvbfd);
        hipFree(dvbfd2);
        hipFree(dvbffcpd);
        hipFree(Ldev);
        hipFree(Ha);
        hipFree(Hb);
        
    }

    else {
        cout << "データファイルを開けませんでした\n終了します。";

    }

    return 0;
}